#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
    return result;
}

__global__ void kernel(float *a, float *b, const int n)
{
    float u = -0.50;
    float t = 1.0;
    float h = 0.99;

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ((i + 1) < n)
    {
        a[i] += ((u * t) / h) * (a[i + 1] - a[i]); 
    }
}

float init_function(float x) 
{
    return sin(x * 0.02);
}

int main(int argc, char **argv)
{
    const int num_of_blocks     = 10;
    const int threads_per_block = 32;
    const int vector_size       = num_of_blocks * threads_per_block;
    const int bytes             = vector_size * sizeof(float);

    int devId = 0;
    hipDeviceProp_t properties;
    hipEvent_t    start_event;
    hipEvent_t    stop_event;

    float *host_vector1   = NULL;
    float *host_vector2   = NULL;
    float *device_vector1 = NULL;
    float *device_vector2 = NULL;

    float ms = 0.0; // Elapsed time in milliseconds

    checkCuda(hipGetDeviceProperties(&properties, devId));
    printf("Device : %s\n", properties.name);
    checkCuda(hipSetDevice(devId));

    // Allocate memory for the host vector
    host_vector1 = (float *)malloc(bytes);
    if (!host_vector1)
    {
        fprintf(stderr, "Can't allocate memory for vector\n");
        exit(EXIT_FAILURE);
    }
    
    host_vector2 = (float *)malloc(bytes);
    if (!host_vector1)
    {
        fprintf(stderr, "Can't allocate memory for vector\n");
        exit(EXIT_FAILURE);
    }

    // Initialize host vector
    for (int i = 0; i < vector_size; i++)
    {
        host_vector1[i] = init_function((float)i);
        host_vector2[i] = init_function((float)i);
    }

    // Allocate memory for the device vector
    checkCuda(checkCuda(hipMalloc((void**)&device_vector1, bytes)));
    checkCuda(checkCuda(hipMalloc((void**)&device_vector2, bytes)));

    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    // Run it
    checkCuda(hipEventRecord(start_event, 0));
    for (int i = 0; i < 50; i++)
    {
        // Copy to device (gpu)
        checkCuda(hipMemcpy(device_vector1, host_vector1, bytes, hipMemcpyHostToDevice));
        checkCuda(hipMemcpy(device_vector2, host_vector2, bytes, hipMemcpyHostToDevice));
        kernel<<<dim3(num_of_blocks), dim3(threads_per_block)>>>(device_vector1, device_vector2, vector_size);
        checkCuda(hipDeviceSynchronize());
        // Copy from device (gpu)
        checkCuda(hipMemcpy(host_vector1, device_vector1, bytes, hipMemcpyDeviceToHost));
        checkCuda(hipMemcpy(host_vector2, device_vector2, bytes, hipMemcpyDeviceToHost));

        char name[32] = { 0 };
        sprintf(name, "sin-cuda-%d.txt", i);
        FILE *fp = fopen(name, "w+");
        for (int j = 0; j < vector_size; j++)
        {
            //printf("%.6f\t%.6f\t%.6f\n", (float)j * 0.02, host_vector1[j], host_vector2[j]);
            fprintf(fp, "%.6f\t%.6f\n", (float)j * 0.02, host_vector1[j]);
            //host_vector1[j] = host_vector2[j];
        }
        //printf("\n");
        fclose(fp);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&ms, start_event, stop_event);

    printf("Running cuda-sin implementation\n");
    printf("Elapsed time: %f\n", ms);

    checkCuda(hipFree(device_vector1));
    checkCuda(hipFree(device_vector2));
    free(host_vector1);
    free(host_vector2);

    return 0;
}
