#include <hip/hip_runtime.h>
#include <stdio.h>

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
    return result;
}

__global__ void kernel(float *a, float *b, int offset)
{
    int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
    a[i] = a[i] + b[i];
}

float validation(float *a, int n) 
{
    float result = 0.0;
    for (int i = 0; i < n; i++)
    {
        result += a[i];
    }
    return result;
}

int main(int argc, char **argv)
{
    const int blockSize   = 512;
    const int nStreams    = 8;
    const int n           = 2 * 1024 * blockSize * nStreams;
    const int streamSize  = n / nStreams;
    const int streamBytes = streamSize * sizeof(float);
    const int bytes       = n * sizeof(float);

    int devId = 0;

    hipDeviceProp_t prop;

    checkCuda( hipGetDeviceProperties(&prop, devId));
    printf("Device : %s\n", prop.name);
    checkCuda( hipSetDevice(devId) );

    // Allocate pinned host memory and device memory
    float *array = NULL;
    float *a     = NULL;
    float *b     = NULL;
    float *d_a   = NULL;
    float *d_b   = NULL;

    checkCuda(hipHostMalloc((void**)&a, bytes, hipHostMallocDefault)); // Host pinned
    checkCuda(hipHostMalloc((void**)&b, bytes, hipHostMallocDefault)); // Host pinned
    checkCuda(hipMalloc((void**)&d_a, bytes));   // Device
    checkCuda(hipMalloc((void**)&d_b, bytes));   // Device

    array = (float *)malloc(bytes);

    for (int i = 0; i < n; i++)
    {
        array[i] = (float)i;
    }

    float ms = 0.0; // Elapsed time in milliseconds

    // Create events and streams
    hipEvent_t startEvent, stopEvent, dummyEvent;
    hipStream_t stream[nStreams];

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));
    checkCuda(hipEventCreate(&dummyEvent));

    for (int i = 0; i < nStreams; ++i)
    {
        checkCuda(hipStreamCreate(&stream[i]));
    }

    // baseline case - sequential transfer and execute
    memcpy(a, array, bytes);
    memcpy(b, array, bytes);

    checkCuda(hipEventRecord(startEvent,0));
    checkCuda(hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice));

    kernel<<<n / blockSize, blockSize>>>(d_a, d_b, 0);

    checkCuda(hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost));
    checkCuda(hipEventRecord(stopEvent, 0) );
    checkCuda(hipEventSynchronize(stopEvent) );
    checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));

    printf("Time for sequential transfer and execute (ms): %f\n", ms);
    printf("  validation: %e\n", validation(a, n));

    // Asynchronous version: loop over { copy, kernel, copy }
    memcpy(a, array, bytes);
    memcpy(b, array, bytes);

    checkCuda(hipEventRecord(startEvent,0));
    for (int i = 0; i < nStreams; ++i)
    {
        int offset = i * streamSize;

        checkCuda(hipMemcpyAsync(&d_a[offset], &a[offset],
            streamBytes, hipMemcpyHostToDevice, stream[i]));

        checkCuda(hipMemcpyAsync(&d_b[offset], &b[offset],
            streamBytes, hipMemcpyHostToDevice, stream[i]));
    }

    for (int i = 0; i < nStreams; ++i)
    {
        int offset = i * streamSize;
        kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, d_b, offset);
    }

    for (int i = 0; i < nStreams; ++i)
    {
        int offset = i * streamSize;
        checkCuda(hipMemcpyAsync(&a[offset], &d_a[offset],
            streamBytes, hipMemcpyDeviceToHost, stream[i]));
    }
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("Time for asynchronous transfer and execute (ms): %f\n", ms);
    printf("  validation: %e\n", validation(a, n));

    for (int i = 0; i < n; i++)
    {
        array[i] = array[i] + array[i];
    }

    printf("  validation: %e\n", validation(a, n));

    // Cleanup
    checkCuda(hipEventDestroy(startEvent));
    checkCuda(hipEventDestroy(stopEvent));
    checkCuda(hipEventDestroy(dummyEvent));

    for (int i = 0; i < nStreams; ++i)
    {
        checkCuda( hipStreamDestroy(stream[i]));
    }

    hipFree(d_a);
    hipFree(d_b);
    hipHostFree(a);
    hipHostFree(b);

    free(array);

    return 0;
}
